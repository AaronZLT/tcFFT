#include "hip/hip_runtime.h"
#include "tcfft_half_2d.h"
using namespace nvcuda;
const int WARP_SIZE = 32, WMMA_M = 16, WMMA_N = 16, WMMA_K = 16, CONT_SIZE = 32;

__device__ inline void complex_mul(wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> &frag_F_real, wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> &frag_F_imag,
                                   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> &frag_in_real, wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> &frag_in_imag,
                                   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> &frag_out_real, wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> &frag_out_imag)
{
    wmma::fill_fragment(frag_out_real, 0.0);
    wmma::fill_fragment(frag_out_imag, 0.0);

    wmma::mma_sync(frag_out_real, frag_F_imag, frag_in_imag, frag_out_real);
    for (int i = 0; i < frag_out_real.num_elements; i++)
        frag_out_real.x[i] = -frag_out_real.x[i];
    wmma::mma_sync(frag_out_real, frag_F_real, frag_in_real, frag_out_real);

    wmma::mma_sync(frag_out_imag, frag_F_real, frag_in_imag, frag_out_imag);
    wmma::mma_sync(frag_out_imag, frag_F_imag, frag_in_real, frag_out_imag);
}

__device__ inline void complex_mul_acc(wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> &frag_F_real, wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> &frag_F_imag,
                                       wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> &frag_in_real, wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> &frag_in_imag,
                                       wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> &frag_out_real, wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> &frag_out_imag)
{
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_buf_real;
    wmma::fill_fragment(frag_buf_real, 0.0);

    wmma::mma_sync(frag_buf_real, frag_F_imag, frag_in_imag, frag_buf_real);
    for (int i = 0; i < frag_buf_real.num_elements; i++)
        frag_buf_real.x[i] = -frag_buf_real.x[i];
    wmma::mma_sync(frag_buf_real, frag_F_real, frag_in_real, frag_buf_real);
    for (int i = 0; i < frag_buf_real.num_elements; i++)
        frag_out_real.x[i] += frag_buf_real.x[i];

    wmma::mma_sync(frag_out_imag, frag_F_real, frag_in_imag, frag_out_imag);
    wmma::mma_sync(frag_out_imag, frag_F_imag, frag_in_real, frag_out_imag);
}

__device__ __host__ inline half2 W_N_K(int N, int K)
{
    half2 t = {cosf(2 * M_PI * K / N), -sinf(2 * M_PI * K / N)};
    return t;
}

__device__ __host__ inline float2 W_N_K_fp32(int N, int K)
{
    float2 t = {cosf(2 * M_PI * K / N), -sinf(2 * M_PI * K / N)};
    return t;
}

__device__ inline half2 const cmul(const half2 &a, const half2 &b)
{
    return {a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}

__device__ inline half2 const cmul_mixed(const half2 &a, const float2 &b)
{
    return {a.x * __float2half(b.x) - a.y * __float2half(b.y), a.x * __float2half(b.y) + a.y * __float2half(b.x)};
}

__device__ inline void swap(half &a, half &b)
{
    half tmp = a;
    a = b;
    b = tmp;
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_256_0(half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.x * 256 * CONT_SIZE;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    half2 twiddle_unit = W_N_K(256, raw_col);

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_tmp0;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_tmp1;

        int warp_start = i + threadIdx.y * 256;
        wmma::load_matrix_sync(frag_in_tmp0, (half *)(in + block_start + warp_start), 32);
        wmma::load_matrix_sync(frag_in_tmp1, (half *)(in + block_start + warp_start) + 16, 32);

        for (int j = 0; j < 8; ++j)
        {
            frag_in_real.x[j] = frag_in_tmp0.x[2 * j];
            frag_in_imag.x[j] = frag_in_tmp0.x[2 * j + 1];
            frag_in_real.x[8 + j] = frag_in_tmp1.x[2 * j];
            frag_in_imag.x[8 + j] = frag_in_tmp1.x[2 * j + 1];
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        wmma::store_matrix_sync((half *)(smem_in + warp_start), frag_out_real, 16, wmma::mem_row_major);
        wmma::store_matrix_sync((half *)(smem_in + warp_start) + 256, frag_out_imag, 16, wmma::mem_row_major);
    /* opt test
    }
    __syncthreads();
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid / 32 * 32 + eid % 32 / 2 + eid % 32 % 2 * 16] = smem_in[eid];
    }
    __syncthreads();

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        int warp_start = i + threadIdx.y * 256;
    */

        wmma::load_matrix_sync(frag_in_real, (half *)(smem_in + warp_start), 16);
        wmma::load_matrix_sync(frag_in_imag, (half *)(smem_in + warp_start) + 256, 16);

        half2 twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            int row = j;
            int col = raw_col;
            half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
            in_ele = cmul(in_ele, twiddle_factor);
            frag_in_real.x[j] = in_ele.x;
            frag_in_imag.x[j] = in_ele.y;
            twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int raw_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        raw_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row;
            int col = j + raw_col;
            smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        in[block_start + eid] = smem_in[eid];
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_256_0_A100(half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.x * 256 * CONT_SIZE;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_row = threadIdx.x % 4 * 2;
    int raw_col = threadIdx.x / 4;
    // half2 twiddle_unit = W_N_K(256, raw_col);

    /* opt test
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid] = in[block_start + eid];
    }
    __syncthreads();

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid / 32 * 32 + eid % 32 / 2 + eid % 32 % 2 * 16] = smem_in[eid];
    }
    __syncthreads();
    */

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;

        int warp_start = i + threadIdx.y * 256;

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 ele = in[block_start + warp_start + row + col * 16];
            // half2 ele = smem_in[warp_start + row + col * 16]; // opt test
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        wmma::store_matrix_sync((half *)(smem_in + warp_start), frag_out_real, 16, wmma::mem_row_major);
        wmma::store_matrix_sync((half *)(smem_in + warp_start) + 256, frag_out_imag, 16, wmma::mem_row_major);

        wmma::load_matrix_sync(frag_in_real, (half *)(smem_in + warp_start), 16);
        wmma::load_matrix_sync(frag_in_imag, (half *)(smem_in + warp_start) + 256, 16);

        // half2 twiddle_factor = {1.0, 0};
        // for (int j = 0; j < 16; ++j)
        // {
        //     int row = j;
        //     int col = raw_col;
        //     half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
        //     in_ele = cmul(in_ele, twiddle_factor);
        //     frag_in_real.x[j] = in_ele.x;
        //     frag_in_imag.x[j] = in_ele.y;
        //     twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        // }
        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
            in_ele = cmul(in_ele, W_N_K(256, row * col));
            frag_in_real.x[8 + j] = frag_in_real.x[j] = in_ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = in_ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        // int raw_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        // raw_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            in[block_start + warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
            // smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]}; //opt test
        }
    }

    /* opt test
    __syncthreads();
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        in[block_start + eid] = smem_in[eid];
    }
    */
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_256_1(int step, half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.y * step * 256 + blockIdx.x * CONT_SIZE;

    int b_c_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    // int glb_col = blockIdx.x * CONT_SIZE + threadIdx.y % 2 * 16 + b_c_col;
    // half2 twiddle_unit = W_N_K(step * 16, glb_col);

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    /* opt test
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid] = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
    }
    __syncthreads();

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid / 32 * 32 + eid % 32 / 2 + eid % 32 % 2 * 16] = smem_in[eid];
    }
    __syncthreads();
    */

    for (int i_start = 0; i_start < 256 * CONT_SIZE; i_start += NUM_WARP * 256)
    {
        int warp_start = i_start + threadIdx.y / 2 * 512 + threadIdx.y % 2 * 16;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        // half2 twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            int col = b_c_col;
            int row = j;
            int eid = warp_start + row * 32 + col;
            half2 ele = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
            // ele = cmul(ele, twiddle_factor);
            frag_in_real.x[j] = ele.x;
            frag_in_imag.x[j] = ele.y;
            // twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int acc_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        int acc_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = acc_row;
            int col = j + acc_col;
            smem_in[warp_start + row * 32 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();
    /* opt test
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid / 32 * 32 + eid % 32 / 2 + eid % 32 % 2 * 16] = smem_in[eid];
    }
    __syncthreads();
    */

    for (int i_start = 0; i_start < CONT_SIZE / NUM_WARP; i_start++)
    {
        int warp_start = i_start * NUM_WARP * 16 + threadIdx.y * 16;
        // int glb_col_2 = blockIdx.x * CONT_SIZE + i_start * step * 4 + threadIdx.y / 2 * step + threadIdx.y % 2 * 16 + b_c_col;
        int glb_col_2 = i_start * 4 + threadIdx.y / 2;
        // half2 twiddle_unit_2 = W_N_K(step * 256, glb_col_2);
        half2 twiddle_unit_2 = W_N_K(256, glb_col_2);
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        half2 twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            int col = b_c_col;
            int row = j;
            half2 ele = smem_in[warp_start + row * 512 + col];
            ele = cmul(ele, twiddle_factor);
            frag_in_real.x[j] = ele.x;
            frag_in_imag.x[j] = ele.y;
            twiddle_factor = cmul(twiddle_factor, twiddle_unit_2);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int acc_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        int acc_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = acc_row;
            int col = j + acc_col;
            smem_in[warp_start + row * 512 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE] = smem_in[eid];
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_256_1_A100(int step, half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.y * step * 256 + blockIdx.x * CONT_SIZE;

    // int b_c_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    // int glb_col = blockIdx.x * CONT_SIZE + threadIdx.y % 2 * 16 + b_c_col;
    // half2 twiddle_unit = W_N_K(step * 16, glb_col);
    int warp_col = blockIdx.x * CONT_SIZE + threadIdx.y % 2 * 16;

    int raw_row = threadIdx.x % 4 * 2;
    int raw_col = threadIdx.x / 4;

    half2 twiddle_factor;
    half2 twiddle_unit;
    // for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    // {
    //     int eid = i + t_block;
    //     smem_in[eid] = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
    // }

    for (int i = 0; i < 2; ++i)
    {
        int eid = i * 512 * 8 + threadIdx.y * 512 + threadIdx.x;
        for (int j = 0; j < 16; ++j)
        {
            smem_in[eid] = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
            eid += 32;
        }
    }

    __syncthreads();

    /* opt test
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid / 32 * 32 + eid % 32 / 2 + eid % 32 % 2 * 16] = smem_in[eid];
    }
    __syncthreads();
    */

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    half2 twiddle[8];
    // for (int j = 0; j < 8; ++j)
    // {
    //     int row = raw_row + j % 4 / 2 * 8 + j % 2;
    //     int col = raw_col + j / 4 * 8;
    //     twiddle[j] = W_N_K(step * 16, (warp_col + col) * row);
    // }

    for (int i_start = 0; i_start < 256 * CONT_SIZE; i_start += NUM_WARP * 256)
    {
        int warp_start = i_start + threadIdx.y / 2 * 512 + threadIdx.y % 2 * 16;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            int eid = warp_start + row * 32 + col;
            half2 ele = smem_in[eid];
            // ele = cmul(ele, twiddle[j]);
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
        }

        // half2 twiddle_factor = {1.0, 0};
        // for (int j = 0; j < 16; ++j)
        // {
        //     int col = b_c_col;
        //     int row = j;
        //     int eid = warp_start + row * 32 + col;
        //     half2 ele = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
        //     ele = cmul(ele, twiddle_factor);
        //     frag_in_real.x[j] = ele.x;
        //     frag_in_imag.x[j] = ele.y;
        //     twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        // }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            smem_in[warp_start + row * 32 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    // warp_col = blockIdx.x * CONT_SIZE + threadIdx.y / 2 * step + threadIdx.y % 2 * 16;
    // for (int j = 0; j < 8; ++j)
    // {
    //     int row = raw_row + j % 4 / 2 * 8 + j % 2;
    //     int col = raw_col + j / 4 * 8;
    //     twiddle[j] = W_N_K(step * 256, warp_col * row + col * row);
    // }
    // half2 twiddle_unit_2[4];
    // for (int j = 0; j < 4; ++j)
    // {
    //     int row = raw_row + j / 2 * 8 + j % 2;
    //     twiddle_unit_2[j] = W_N_K(step * 256, step * 4 * row);
    // }

    for (int i = 0; i < 2; ++i)
    {
        twiddle_unit = W_N_K(256, threadIdx.y + i * 8);
        int eid = i * 32 * 8 + threadIdx.y * 32 + threadIdx.x;
        twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            smem_in[eid] = cmul(smem_in[eid], twiddle_factor);
            eid += 512;
            twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }
    }

    __syncthreads();

    for (int i_start = 0; i_start < CONT_SIZE / NUM_WARP; i_start++)
    {
        int warp_start = i_start * NUM_WARP * 16 + threadIdx.y * 16;
        // int glb_col_2 = blockIdx.x * CONT_SIZE + c + threadIdx.y / 2 * step + threadIdx.y % 2 * 16 + b_c_col;
        // half2 twiddle_unit_2 = W_N_K(step * 256, glb_col_2);
        // warp_col = blockIdx.x * CONT_SIZE + i_start * step * 4 + threadIdx.y / 2 * step + threadIdx.y % 2 * 16;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        // half2 twiddle_factor = {1.0, 0};
        // for (int j = 0; j < 16; ++j)
        // {
        //     int col = b_c_col;
        //     int row = j;
        //     half2 ele = smem_in[warp_start + row * 512 + col];
        //     ele = cmul(ele, twiddle_factor);
        //     frag_in_real.x[j] = ele.x;
        //     frag_in_imag.x[j] = ele.y;
        //     twiddle_factor = cmul(twiddle_factor, twiddle_unit_2);
        // }

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 ele = smem_in[warp_start + row * 512 + col];
            // ele = cmul(ele, twiddle[j]);
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
            // twiddle[j] = cmul(twiddle[j], twiddle_unit_2[j % 4]);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            smem_in[warp_start + row * 512 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    /* opt test
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        smem_in[eid / 32 * 32 + eid % 32 / 2 + eid % 32 % 2 * 16] = smem_in[eid];
    }
    __syncthreads();
    */

    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE] = smem_in[eid];
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_512_0(half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.x * 512 * CONT_SIZE;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    half2 twiddle_unit = W_N_K(256, raw_col);
    half2 twiddle_two = W_N_K(512, t_block);

    for (int i = 0; i < 512 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_tmp0;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_tmp1;

        int warp_start = i + threadIdx.y * 256;
        wmma::load_matrix_sync(frag_in_tmp0, (half *)(in + block_start + warp_start), 32);
        wmma::load_matrix_sync(frag_in_tmp1, (half *)(in + block_start + warp_start) + 16, 32);

        for (int j = 0; j < 8; ++j)
        {
            frag_in_real.x[j] = frag_in_tmp0.x[2 * j];
            frag_in_imag.x[j] = frag_in_tmp0.x[2 * j + 1];
            frag_in_real.x[8 + j] = frag_in_tmp1.x[2 * j];
            frag_in_imag.x[8 + j] = frag_in_tmp1.x[2 * j + 1];
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        wmma::store_matrix_sync((half *)(smem_in + warp_start), frag_out_real, 16, wmma::mem_row_major);
        wmma::store_matrix_sync((half *)(smem_in + warp_start) + 256, frag_out_imag, 16, wmma::mem_row_major);

        wmma::load_matrix_sync(frag_in_real, (half *)(smem_in + warp_start), 16);
        wmma::load_matrix_sync(frag_in_imag, (half *)(smem_in + warp_start) + 256, 16);

        half2 twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            int row = j;
            int col = raw_col;
            half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
            in_ele = cmul(in_ele, twiddle_factor);
            frag_in_real.x[j] = in_ele.x;
            frag_in_imag.x[j] = in_ele.y;
            twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int raw_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        raw_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row;
            int col = j + raw_col;
            smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();
    for (int i = 0; i < 512 * CONT_SIZE; i += NUM_WARP * 32 * 2)
    {
        int eid = i + t_block;
        half2 ele_0 = smem_in[eid];
        half2 ele_1 = cmul(smem_in[eid + 256], twiddle_two);
        in[block_start + eid] = __hadd2(ele_0, ele_1);
        in[block_start + eid + 256] = __hsub2(ele_0, ele_1);
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_512_0_A100(half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.x * 512 * CONT_SIZE;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_row = threadIdx.x % 4 * 2;
    int raw_col = threadIdx.x / 4;
    half2 twiddle_two = W_N_K(512, t_block);

    for (int i = 0; i < 512 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;

        int warp_start = i + threadIdx.y * 256;

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 ele = in[block_start + warp_start + row + col * 16];
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        wmma::store_matrix_sync((half *)(smem_in + warp_start), frag_out_real, 16, wmma::mem_row_major);
        wmma::store_matrix_sync((half *)(smem_in + warp_start) + 256, frag_out_imag, 16, wmma::mem_row_major);

        wmma::load_matrix_sync(frag_in_real, (half *)(smem_in + warp_start), 16);
        wmma::load_matrix_sync(frag_in_imag, (half *)(smem_in + warp_start) + 256, 16);

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
            in_ele = cmul(in_ele, W_N_K(256, row * col));
            frag_in_real.x[8 + j] = frag_in_real.x[j] = in_ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = in_ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();
    for (int i = 0; i < 512 * CONT_SIZE; i += NUM_WARP * 32 * 2)
    {
        int eid = i + t_block;
        half2 ele_0 = smem_in[eid];
        half2 ele_1 = cmul(smem_in[eid + 256], twiddle_two);
        in[block_start + eid] = __hadd2(ele_0, ele_1);
        in[block_start + eid + 256] = __hsub2(ele_0, ele_1);
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_512_1(int step, half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.y * step * 512 + blockIdx.x * CONT_SIZE;

    int b_c_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    // int glb_col = blockIdx.x * CONT_SIZE + b_c_col;
    // half2 twiddle_unit = W_N_K(step * 16, glb_col);

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    for (int i_start = 0; i_start < 512 * CONT_SIZE; i_start += NUM_WARP * 256)
    {
        int warp_start = i_start + threadIdx.y * 256;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        // half2 twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            int col = b_c_col;
            int row = j;
            int eid = warp_start + row * CONT_SIZE + col;
            half2 ele = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
            // ele = cmul(ele, twiddle_factor);
            frag_in_real.x[j] = ele.x;
            frag_in_imag.x[j] = ele.y;
            // twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int acc_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        int acc_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = acc_row;
            int col = j + acc_col;
            smem_in[warp_start + row * CONT_SIZE + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    for (int i_start = 0; i_start < 4; i_start++)
    {
        int warp_start = i_start % 2 * NUM_WARP * 16 + i_start / 2 * 256 * CONT_SIZE + threadIdx.y * 16;
        // int glb_col_2 = blockIdx.x * CONT_SIZE + i_start % 2 * step * 8 + threadIdx.y * step + b_c_col;
        int glb_col_2 = i_start % 2 * 8 + threadIdx.y;
        // half2 twiddle_unit_2 = W_N_K(step * 256, glb_col_2);
        // half2 twiddle_unit_2 = W_N_K(256, glb_col_2);
        float2 twiddle_unit_2 = W_N_K_fp32(256, glb_col_2); // precision improved
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        // half2 twiddle_factor = {1.0, 0};
        float2 twiddle_factor = {1.0, 0}; // precision improved
        for (int j = 0; j < 16; ++j)
        {
            int col = b_c_col;
            int row = j;
            half2 ele = smem_in[warp_start + row * 256 + col];
            // ele = cmul(ele, twiddle_factor);
            ele = cmul_mixed(ele, twiddle_factor); // precision improved
            frag_in_real.x[j] = ele.x;
            frag_in_imag.x[j] = ele.y;
            // twiddle_factor = cmul(twiddle_factor, twiddle_unit_2);
            twiddle_factor = hipCmulf(twiddle_factor, twiddle_unit_2); // precision improved
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int acc_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        int acc_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = acc_row;
            int col = j + acc_col;
            smem_in[warp_start + row * 256 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    // half2 twiddle_unit_2 = W_N_K(step * 512, 256 / CONT_SIZE * step);
    // half2 twiddle_unit_2 = W_N_K(512, 256 / CONT_SIZE);
    float2 twiddle_unit_2 = W_N_K_fp32(512, 256 / CONT_SIZE); // precision improved
    // half2 twiddle_factor = W_N_K(step * 512, t_block / CONT_SIZE * step + t_block % CONT_SIZE);
    // half2 twiddle_factor = W_N_K(512, t_block / CONT_SIZE);
    float2 twiddle_factor = W_N_K_fp32(512, t_block / CONT_SIZE); // precision improved
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        half2 ele_0 = smem_in[eid];
        // half2 ele_1 = cmul(smem_in[eid + 256 * CONT_SIZE], twiddle_factor);
        half2 ele_1 = cmul_mixed(smem_in[eid + 256 * CONT_SIZE], twiddle_factor); // precision improved
        in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE] = __hadd2(ele_0, ele_1);
        eid += 256 * CONT_SIZE;
        in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE] = __hsub2(ele_0, ele_1);
        // twiddle_factor = cmul(twiddle_factor, twiddle_unit_2);
        twiddle_factor = hipCmulf(twiddle_factor, twiddle_unit_2); // precision improved
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_512_1_A100(int step, half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.y * step * 512 + blockIdx.x * CONT_SIZE;

    // int b_c_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    // int glb_col = blockIdx.x * CONT_SIZE + b_c_col;
    // half2 twiddle_unit = W_N_K(step * 16, glb_col);

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_row = threadIdx.x % 4 * 2;
    int raw_col = threadIdx.x / 4;

    half2 twiddle_factor;
    half2 twiddle_unit;

    for (int i = 0; i < 2; ++i)
    {
        int eid = i * 512 * 8 + threadIdx.y * 512 + threadIdx.x / 16 * 256 + threadIdx.x % 16;
        for (int j = 0; j < 16; ++j)
        {
            smem_in[eid] = in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE];
            eid += 16;
        }
    }

    __syncthreads();

    for (int i_start = 0; i_start < 512 * CONT_SIZE; i_start += NUM_WARP * 256)
    {
        int warp_start = i_start + threadIdx.y * 256;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            int eid = warp_start + row * 16 + col;
            half2 ele = smem_in[eid];
            // ele = cmul(ele, twiddle[j]);
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    for (int i = 0; i < 2; ++i)
    {
        twiddle_unit = W_N_K(256, threadIdx.y * 2 + threadIdx.x / 16);
        int eid = i * 16 * 16 * 16 + threadIdx.y * 32 + threadIdx.x;
        twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            smem_in[eid] = cmul(smem_in[eid], twiddle_factor);
            eid += 256;
            twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }
    }

    __syncthreads();

    for (int i_start = 0; i_start < 4; i_start++)
    {
        int warp_start = i_start % 2 * NUM_WARP * 16 + i_start / 2 * 256 * CONT_SIZE + threadIdx.y * 16;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 ele = smem_in[warp_start + row * 256 + col];
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            smem_in[warp_start + row * 256 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    __syncthreads();

    half2 twiddle_unit_2 = W_N_K(512, 256 / CONT_SIZE);
    twiddle_factor = W_N_K(512, t_block / CONT_SIZE);
    for (int i = 0; i < 256 * CONT_SIZE; i += NUM_WARP * 32)
    {
        int eid = i + t_block;
        half2 ele_0 = smem_in[eid];
        half2 ele_1 = cmul(smem_in[eid + 256 * CONT_SIZE], twiddle_factor);
        in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE] = __hadd2(ele_0, ele_1);
        eid += 256 * CONT_SIZE;
        in[block_start + eid / CONT_SIZE * step + eid % CONT_SIZE] = __hsub2(ele_0, ele_1);
        twiddle_factor = cmul(twiddle_factor, twiddle_unit_2);
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_1024_0(half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.x * 1024 * CONT_SIZE;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_col = threadIdx.x / 16 * 4 + threadIdx.x % 16 / 8 * 8 + threadIdx.x % 4;
    half2 twiddle_unit = W_N_K(256, raw_col);

    for (int i = 0; i < 1024 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_tmp0;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_tmp1;

        int warp_start = i + threadIdx.y * 256;
        wmma::load_matrix_sync(frag_in_tmp0, (half *)(in + block_start + warp_start), 32);
        wmma::load_matrix_sync(frag_in_tmp1, (half *)(in + block_start + warp_start) + 16, 32);

        for (int j = 0; j < 8; ++j)
        {
            frag_in_real.x[j] = frag_in_tmp0.x[2 * j];
            frag_in_imag.x[j] = frag_in_tmp0.x[2 * j + 1];
            frag_in_real.x[8 + j] = frag_in_tmp1.x[2 * j];
            frag_in_imag.x[8 + j] = frag_in_tmp1.x[2 * j + 1];
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        wmma::store_matrix_sync((half *)(smem_in + warp_start), frag_out_real, 16, wmma::mem_row_major);
        wmma::store_matrix_sync((half *)(smem_in + warp_start) + 256, frag_out_imag, 16, wmma::mem_row_major);

        wmma::load_matrix_sync(frag_in_real, (half *)(smem_in + warp_start), 16);
        wmma::load_matrix_sync(frag_in_imag, (half *)(smem_in + warp_start) + 256, 16);

        half2 twiddle_factor = {1.0, 0};
        for (int j = 0; j < 16; ++j)
        {
            int row = j;
            int col = raw_col;
            half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
            in_ele = cmul(in_ele, twiddle_factor);
            frag_in_real.x[j] = in_ele.x;
            frag_in_imag.x[j] = in_ele.y;
            twiddle_factor = cmul(twiddle_factor, twiddle_unit);
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        int raw_row = threadIdx.x / 16 * 4 + threadIdx.x % 8 / 4 * 8 + threadIdx.x % 4;
        raw_col = threadIdx.x % 16 / 8 * 8;
        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row;
            int col = j + raw_col;
            smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    half2 twiddle_1024_1 = W_N_K(1024, t_block);
    half2 twiddle_1024_2 = cmul(twiddle_1024_1, twiddle_1024_1);
    half2 twiddle_1024_3 = cmul(twiddle_1024_2, twiddle_1024_1);

    __syncthreads();
    for (int i = 0; i < 1024 * CONT_SIZE; i += NUM_WARP * 32 * 4)
    {
        int eid = i + t_block;
        half2 ele0 = smem_in[eid];
        half2 ele1 = cmul(smem_in[eid + 256], twiddle_1024_1);
        half2 ele2 = cmul(smem_in[eid + 512], twiddle_1024_2);
        half2 ele3 = cmul(smem_in[eid + 768], twiddle_1024_3);
        in[block_start + eid] = ele0 + ele1 + ele2 + ele3;
        in[block_start + eid + 256] = ele0 + half2({ele1.y, -ele1.x}) - ele2 + half2({-ele3.y, ele3.x});
        in[block_start + eid + 512] = ele0 - ele1 + ele2 - ele3;
        in[block_start + eid + 768] = ele0 + half2({-ele1.y, ele1.x}) - ele2 + half2({ele3.y, -ele3.x});
    }
}

template <int CONT_SIZE, int NUM_WARP>
__global__ void layer_1024_0_A100(half2 *in, half *F_real, half *F_imag)
{
    extern __shared__ half2 smem_in[];
    int t_block = threadIdx.x + threadIdx.y * blockDim.x;
    int block_start = blockIdx.x * 1024 * CONT_SIZE;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_real;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_F_imag;
    wmma::load_matrix_sync(frag_F_real, F_real, 16);
    wmma::load_matrix_sync(frag_F_imag, F_imag, 16);

    int raw_row = threadIdx.x % 4 * 2;
    int raw_col = threadIdx.x / 4;

    for (int i = 0; i < 1024 * CONT_SIZE; i += NUM_WARP * 16 * 16)
    {
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_real;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_out_imag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_real;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> frag_in_imag;

        int warp_start = i + threadIdx.y * 256;

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 ele = in[block_start + warp_start + row + col * 16];
            frag_in_real.x[8 + j] = frag_in_real.x[j] = ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        wmma::store_matrix_sync((half *)(smem_in + warp_start), frag_out_real, 16, wmma::mem_row_major);
        wmma::store_matrix_sync((half *)(smem_in + warp_start) + 256, frag_out_imag, 16, wmma::mem_row_major);

        wmma::load_matrix_sync(frag_in_real, (half *)(smem_in + warp_start), 16);
        wmma::load_matrix_sync(frag_in_imag, (half *)(smem_in + warp_start) + 256, 16);

        for (int j = 0; j < 8; ++j)
        {
            int row = raw_row + j % 4 / 2 * 8 + j % 2;
            int col = raw_col + j / 4 * 8;
            half2 in_ele = {frag_in_real.x[j], frag_in_imag.x[j]};
            in_ele = cmul(in_ele, W_N_K(256, row * col));
            frag_in_real.x[8 + j] = frag_in_real.x[j] = in_ele.x;
            frag_in_imag.x[8 + j] = frag_in_imag.x[j] = in_ele.y;
        }

        complex_mul(frag_F_real, frag_F_imag, frag_in_real, frag_in_imag, frag_out_real, frag_out_imag);

        for (int j = 0; j < 8; ++j)
        {
            int col = raw_row + j / 4 * 8 + j % 2;
            int row = raw_col + j % 4 / 2 * 8;
            smem_in[warp_start + row * 16 + col] = {frag_out_real.x[j], frag_out_imag.x[j]};
        }
    }

    half2 twiddle_1024_1 = W_N_K(1024, t_block);
    half2 twiddle_1024_2 = cmul(twiddle_1024_1, twiddle_1024_1);
    half2 twiddle_1024_3 = cmul(twiddle_1024_2, twiddle_1024_1);

    __syncthreads();
    for (int i = 0; i < 1024 * CONT_SIZE; i += NUM_WARP * 32 * 4)
    {
        int eid = i + t_block;
        half2 ele0 = smem_in[eid];
        half2 ele1 = cmul(smem_in[eid + 256], twiddle_1024_1);
        half2 ele2 = cmul(smem_in[eid + 512], twiddle_1024_2);
        half2 ele3 = cmul(smem_in[eid + 768], twiddle_1024_3);
        in[block_start + eid] = ele0 + ele1 + ele2 + ele3;
        in[block_start + eid + 256] = ele0 + half2({ele1.y, -ele1.x}) - ele2 + half2({-ele3.y, ele3.x});
        in[block_start + eid + 512] = ele0 - ele1 + ele2 - ele3;
        in[block_start + eid + 768] = ele0 + half2({-ele1.y, ele1.x}) - ele2 + half2({ele3.y, -ele3.x});
    }
}

void tcfftExec(tcfftHandle plan, half *data)
{
    const int num_warp = 8;
    const int n_cont[3] = {32, 16, 8};

    int step = 1;
    int RADIX = 1;
    dim3 threads, blocks;

    // V100
    RADIX = plan.Ny;
    threads = {32, num_warp};
    hipFuncSetAttribute(reinterpret_cast<const void*>(plan.layer_0[plan.mergings[0]]), hipFuncAttributeMaxDynamicSharedMemorySize, RADIX * sizeof(half2) * n_cont[plan.mergings[0]]);
    plan.layer_0[plan.mergings[0]]<<<plan.Nx * plan.Ny * plan.N_batch / n_cont[plan.mergings[0]] / RADIX, threads, RADIX * sizeof(half2) * n_cont[plan.mergings[0]]>>>((half2 *)data, plan.F_real, plan.F_imag);
    step *= RADIX;

    RADIX = plan.Nx;
    blocks = {step / n_cont[plan.mergings[1]], plan.N_batch * plan.Nx * plan.Ny / step / RADIX};
    hipFuncSetAttribute(reinterpret_cast<const void*>(plan.layer_1[plan.mergings[1]]), hipFuncAttributeMaxDynamicSharedMemorySize, RADIX * sizeof(half2) * n_cont[plan.mergings[1]]);
    plan.layer_1[plan.mergings[1]]<<<blocks, threads, RADIX * sizeof(half2) * n_cont[plan.mergings[1]]>>>(step, (half2 *)data, plan.F_real, plan.F_imag);
    step *= RADIX;
}

void tcfftCreate(tcfftHandle *plan, int nx, int ny, int n_batch)
{
    plan->Nx = nx;
    plan->Ny = ny;
    plan->N_batch = n_batch;
    // setup functions
    const int num_warp = 8;
    const int n_cont_256 = 32;
    const int n_cont_512 = 16;
    const int n_cont_1024 = 8;
    plan->layer_0[0] = layer_256_0<n_cont_256, num_warp>;
    plan->layer_0[1] = layer_512_0<n_cont_512, num_warp>;
    plan->layer_0[2] = layer_1024_0<n_cont_1024, num_warp>;
    plan->layer_1[0] = layer_256_1<n_cont_256, num_warp>;
    plan->layer_1[1] = layer_512_1<n_cont_512, num_warp>;
    // radices
    switch (nx)
    {
    case 256:
        plan->n_radices_x = 2;
        break;

    case 512:
        plan->n_radices_x = 3;
        plan->mergings[1] = 1;
        break;

    case 1024:
        plan->n_radices_x = 3;
        plan->radices_x[2] = 4;
        plan->mergings[1] = 2;
        break;

    default:
        break;
    }
    switch (ny)
    {
    case 256:
        plan->n_radices_y = 2;
        break;

    case 512:
        plan->n_radices_y = 3;
        plan->mergings[0] = 1;
        break;

    case 1024:
        plan->n_radices_y = 3;
        plan->radices_y[2] = 4;
        plan->mergings[0] = 2;
        break;

    default:
        break;
    }
    // F
    plan->F_real_tmp = (half *)malloc(sizeof(half) * 256);
    plan->F_imag_tmp = (half *)malloc(sizeof(half) * 256);
#pragma omp parallel for
    for (int i = 0; i < 16; ++i)
        for (int j = 0; j < 16; ++j)
        {
            plan->F_real_tmp[16 * i + j] = cosf(2 * M_PI * i * j / 16);
            plan->F_imag_tmp[16 * i + j] = -sinf(2 * M_PI * i * j / 16);
        }
    hipMalloc(&plan->F_real, sizeof(half) * 256);
    hipMemcpy(plan->F_real, plan->F_real_tmp, sizeof(half) * 256, hipMemcpyHostToDevice);
    hipMalloc(&plan->F_imag, sizeof(half) * 256);
    hipMemcpy(plan->F_imag, plan->F_imag_tmp, sizeof(half) * 256, hipMemcpyHostToDevice);
}